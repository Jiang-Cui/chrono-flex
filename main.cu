#include "hip/hip_runtime.h"
#include "include.cuh"

bool updateDraw = 1;
bool showSphere = 1;

ANCFSystem sys;
OpenGLCamera oglcamera(camreal3(0,0,-1),camreal3(0,0,0),camreal3(0,1,0),1);

//RENDERING STUFF
void changeSize(int w, int h) {
	if(h == 0) {h = 1;}
	float ratio = 1.0* w / h;
	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	glViewport(0, 0, w, h);
	gluPerspective(45,ratio,.1,1000);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	gluLookAt(0.0,0.0,0.0,		0.0,0.0,-7,		0.0f,1.0f,0.0f);
}

void initScene(){
	GLfloat light_position[] = { 1.0, 1.0, 1.0, 0.0 };
	glClearColor (1.0, 1.0, 1.0, 0.0);
	glShadeModel (GL_SMOOTH);
	glEnable(GL_COLOR_MATERIAL);
	glLightfv(GL_LIGHT0, GL_POSITION, light_position);
	glEnable(GL_LIGHTING);
	glEnable(GL_LIGHT0);
	glEnable (GL_POINT_SMOOTH);
	glEnable (GL_BLEND);
	glBlendFunc (GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
	glHint (GL_POINT_SMOOTH_HINT, GL_DONT_CARE);
}

void drawAll()
{
	if(updateDraw){
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
		glEnable(GL_DEPTH_TEST);
		glFrontFace(GL_CCW);
		glCullFace(GL_BACK);
		glEnable(GL_CULL_FACE);
		glDepthFunc(GL_LEQUAL);
		glClearDepth(1.0);

		glPointSize(2);
		glLoadIdentity();

		oglcamera.Update();

//		glColor3f(0.0f,1.0f,0.0f);
//		glBegin(GL_QUADS);
//		double clip =10;
//		glVertex3f(clip,sys.groundHeight,clip);
//		glVertex3f(clip,sys.groundHeight,-clip);
//		glVertex3f(-clip,sys.groundHeight,-clip);
//		glVertex3f(-clip,sys.groundHeight,clip);
//		glEnd();
//		glFlush();

//		glColor3f(0.0f,0.0f,1.0f);
//		glPushMatrix();
//		float3 position = sys.getXYZPosition(100,0);
//		//cout << position.x << " " << position.y << " " << position.z << endl;
//		glTranslatef(position.x,position.y,position.z);
//		glutSolidSphere(1,10,10);
//		glPopMatrix();

		for (int i = 0; i < sys.particles.size(); i++) {
			glColor3f(0.0f, 1.0f, 0.0f);
			glPushMatrix();
			float3 pos = sys.getXYZPositionParticle(i);
			glTranslatef(pos.x, pos.y, pos.z);
			glutSolidSphere(sys.particles[i].getRadius(), 30, 30);
			glPopMatrix();

			//indicate velocity
			glLineWidth(sys.elements[i].getRadius()*500);
			glColor3f(1.0f,0.0f,0.0f);
			glBegin(GL_LINES);
			glVertex3f(pos.x,pos.y,pos.z);
			float3 vel = sys.getXYZVelocityParticle(i);
			//cout << "v:" << vel.x << " " << vel.y << " " << vel.z << endl;
			pos +=2*sys.particles[i].getRadius()*normalize(vel);
			glVertex3f(pos.x,pos.y,pos.z);
			glEnd();
			glFlush();
		}

		for(int i=0;i<sys.elements.size();i++)
		{
			int xiDiv = sys.numContactPoints;

			double xiInc = 1/(static_cast<double>(xiDiv-1));

			if(!showSphere)
			{
				glColor3f(0.0f,0.0f,1.0f);
				for(int j=0;j<xiDiv;j++)
				{
					glPushMatrix();
					float3 position = sys.getXYZPosition(i,xiInc*j);
					glTranslatef(position.x,position.y,position.z);
					glutSolidSphere(sys.elements[i].getRadius(),10,10);
					glPopMatrix();
				}
			}
			else
			{
				int xiDiv = sys.numContactPoints;
				double xiInc = 1/(static_cast<double>(xiDiv-1));
				glLineWidth(sys.elements[i].getRadius()*500);
				glColor3f(0.0f,1.0f,0.0f);
				glBegin(GL_LINE_STRIP);
				for(int j=0;j<sys.numContactPoints;j++)
				{
					float3 position = sys.getXYZPosition(i,xiInc*j);
					glVertex3f(position.x,position.y,position.z);
				}
				glEnd();
				glFlush();
			}
		}

		glutSwapBuffers();
	}
}

void renderSceneAll(){
	if(OGL){
		drawAll();
		sys.DoTimeStep();
	}
}

void CallBackKeyboardFunc(unsigned char key, int x, int y) {
	switch (key) {
	case 'w':
		oglcamera.Forward();
		break;
	case 's':
		oglcamera.Back();
		break;

	case 'd':
		oglcamera.Right();
		break;

	case 'a':
		oglcamera.Left();
		break;

	case 'q':
		oglcamera.Up();
		break;

	case 'e':
		oglcamera.Down();
		break;
	}
}

void CallBackMouseFunc(int button, int state, int x, int y) {
	oglcamera.SetPos(button, state, x, y);
}
void CallBackMotionFunc(int x, int y) {
	oglcamera.Move2D(x, y);
}

int main(int argc, char** argv)
{
//	// begin file reader code
sys.setTimeStep(1e-3);
sys.setTolerance(1e-6);
//	double EM = 2e9;
//	double rho = 2800;
//
//	char *fName = "../data/sphere.ancf";
//
//	int nodeCount = 0;
//	int beamCount = 0;
//	int constraintCount = 0;
//
//	char line[100];
//
//	FILE *fp = fopen(fName, "r");
//
//	vector<float3> nodes;
//	if (fp != NULL) {
//		while (fgets(line, 99, fp)) {
//			if (line[0] == 'n')
//			{
//				float x,y,z;
//				sscanf(line, "%*c %f %f %f", &x,
//						&y, &z);
//				nodeCount++;
//				float3 node = make_float3(x,y,z);
//				nodes.push_back(node);
//				//cout << x << " " << y << " " << z << endl;
//			}
//			else if (line[0] == 'b') {
//				int n1, n2;
//				sscanf(line, "%*c %d %d", &n1, &n2);
//				beamCount++;
//				float3 dir = normalize(nodes[n2-1]-nodes[n1-1]);
//				Element element = Element(Node(nodes[n1-1],dir),Node(nodes[n2-1],dir));
//				element.setRadius(.07);
//				element.setElasticModulus(EM);
//				element.setDensity(rho);
//				sys.addElement(&element);
//			}
//			else if (line[0] == 'c') {
//				int b1, n1, b2, n2;
//				sscanf(line, "%*c %d %d %d %d", &b1, &n1, &b2, &n2);
//				constraintCount++;
//				sys.addConstraint_RelativeSpherical(sys.elements[b1-1],n1-1,sys.elements[b2-1],n2-1);
//			}
//		}
//	}
//	cout << nodeCount << " " << beamCount << " " << constraintCount << endl;
//	sys.addConstraint_AbsoluteSpherical(sys.elements[100],0);
//
//	// end file reader code

	if (argc == 1) {
		Element test = Element();
		//test.setElasticModulus(2e7);
		sys.addElement(&test);
		sys.addConstraint_AbsoluteSpherical(0);
		sys.numContactPoints = 5;
	} else {
		Particle particle1 = Particle(60,70,make_float3(60,70,60),make_float3(0,0,0));
		sys.addParticle(&particle1);
//
//		Particle particle2 = Particle(.6*100,100,make_float3(1200-60,2*100,60),make_float3(0,0,0));
//		sys.addParticle(&particle2);
//
//		Particle particle3 = Particle(.6*100,100,make_float3(1200-60,2*100,1200-60),make_float3(0,0,0));
//		sys.addParticle(&particle3);
//
//		Particle particle4 = Particle(.6*100,100,make_float3(60,2*100,1200-60),make_float3(0,0,0));
//		sys.addParticle(&particle4);

		//Horizontal Net
		sys.setTimeStep(1e-4);
		sys.setTolerance(1e-3);
		sys.detector.setBinsPerAxis(make_uint3(70,10,70));
		//sys.detector.activateDebugMode();
		sys.numContactPoints = 6;
		int numElements = atoi(argv[1]);
		double length = .3*100;
		//double EM = 2e7;
		//double rho = 1150.0;

		Particle particle;
		for( int i=5; i<(numElements-5)/2;i++) {
			for(int j=0;j<5;j++) {
				for(int k=5;k<(numElements-5)/2;k++) {
					particle = Particle(length,10,make_float3(length+2*length*i,3*length+3*length*j,length+2*length*k),make_float3(0,0,0));
					sys.addParticle(&particle);
				}
			}
		}

		Element element;
		int k = 0;
		for (int i = 0; i < numElements+1; i++) {
			for (int j = 0; j < numElements; j++) {
				element = Element(Node(i * length, 0, j * length, 0, 0, 1),
						Node(i * length, 0, (j + 1) * length, 0, 0, 1));
				//element.setElasticModulus(EM);
				//element.setDensity(rho);
				sys.addElement(&element);
				if (sys.elements.size() % 1000 == 0)
					printf("Elements added: %d\n", sys.elements.size());

				//printf("Element #%d: (%f, %f, %f) -> (%f, %f, %f)\n",k,i*length,-j*length,0.0,i*length,-(j+1)*length,0.0);
				k++;
			}
		}

		for (int i = 0; i < numElements+1; i++) {
			if(i==0||i==numElements)
			{
				sys.addConstraint_AbsoluteSpherical(sys.elements[i * numElements], 0);
				sys.addConstraint_AbsoluteSpherical(sys.elements[(i+1)*numElements-1], 1);
				//printf("Constraint: %d\n",numElements*i);
			}
		}

		for (int i = 0; i < numElements+1; i++) {
			for (int j = 0; j < numElements - 1; j++) {
				sys.addConstraint_RelativeFixed(
						sys.elements[j + numElements * i], 1,
						sys.elements[j + 1 + numElements * i], 0);
				//printf("Constraints: %d to %d\n",j+numElements*i,j+1+numElements*i);
			}
		}

		for (int i = 0; i < numElements; i++) {
			element = Element(
					Node(i * length, 0, 0, 1, 0, 0),
					Node((i + 1) * length, 0, 0, 1, 0, 0));
			//element.setElasticModulus(EM);
			//element.setDensity(rho);
			sys.addElement(&element);
			//if (sys.elements.size() % 1000 == 0)
				//printf("Elements added: %d\n", sys.elements.size());

			//printf("Cross Element #%d: (%f, %f, %f) -> (%f, %f, %f)\n",k,i*length,-(j+1)*length,0.0,(i+1)*length,-(j+1)*length,0.0);

			sys.addConstraint_RelativeSpherical(
					sys.elements[0 + numElements * i], 0,
					sys.elements[k], 0);
			sys.addConstraint_RelativeSpherical(
					sys.elements[0 + numElements * (i + 1)], 0,
					sys.elements[k], 1);
			if(i==0) sys.addConstraint_AbsoluteSpherical(sys.elements[k],0);
			if(i==numElements-2) sys.addConstraint_AbsoluteSpherical(sys.elements[k],1);
			//printf("Cross Constraint: %d to %d\n",j+numElements*i,k);
			//printf("Cross Constraint: %d to %d\n",j+numElements*(i+1),k);
			k++;
		}

		for (int i = 0; i < numElements; i++) {
			for (int j = 0; j < numElements; j++) {
				element = Element(
						Node(i * length, 0, (j + 1) * length, 1, 0, 0),
						Node((i + 1) * length, 0, (j + 1) * length, 1, 0, 0));
				//element.setElasticModulus(EM);
				//element.setDensity(rho);
				sys.addElement(&element);
				//if (sys.elements.size() % 1000 == 0)
					//printf("Elements added: %d\n", sys.elements.size());

				//printf("Cross Element #%d: (%f, %f, %f) -> (%f, %f, %f)\n",k,i*length,-(j+1)*length,0.0,(i+1)*length,-(j+1)*length,0.0);

				sys.addConstraint_RelativeSpherical(
						sys.elements[j + numElements * i], 1, sys.elements[k],
						0);
				sys.addConstraint_RelativeSpherical(
						sys.elements[j + numElements * (i + 1)], 1,
						sys.elements[k], 1);
				//printf("Cross Constraint: %d to %d\n",j+numElements*i,k);
				//printf("Cross Constraint: %d to %d\n",j+numElements*(i+1),k);
				if(i==0&&(j==0||j==numElements-1)) sys.addConstraint_AbsoluteSpherical(sys.elements[k],0);
				if(i==numElements-1&&(j==0||j==numElements-1)) sys.addConstraint_AbsoluteSpherical(sys.elements[k],1);
				k++;
			}
		}
	}

//	else
//	{
//		//SOCCER NET
//		sys.setTimeStep(1e-4);
//		sys.setTolerance(1e-8);
//		sys.numContactPoints = 10;
//		int numElements = atoi(argv[1]);
//		double length = .3;
//		double EM = 2e7;
//		double rho = 1150.0;
//
//		Element element;
//		int k = 0;
//		for(int i=0;i<numElements;i++)
//		{
//			for(int j=0;j<numElements;j++)
//			{
//				element = Element(Node(i*length,-j*length,0,0,-1,0),Node(i*length,-(j+1)*length,0,0,-1,0));
//				element.setElasticModulus(EM);
//				element.setDensity(rho);
//				sys.addElement(&element);
//				if(sys.elements.size()%1000==0) printf("Elements added: %d\n",sys.elements.size());
//
//				//printf("Element #%d: (%f, %f, %f) -> (%f, %f, %f)\n",k,i*length,-j*length,0.0,i*length,-(j+1)*length,0.0);
//				k++;
//			}
//		}
//
//		for(int i=0;i<numElements;i++)
//		{
//			sys.addConstraint_AbsoluteSpherical(sys.elements[i*numElements],0);
//			//sys.addConstraint_AbsoluteSpherical(sys.elements[(i+1)*numElements-1],1);
//			//printf("Constraint: %d\n",numElements*i);
//		}
//
//		for(int i=0;i<numElements;i++)
//		{
//			for(int j=0;j<numElements-1;j++)
//			{
//				sys.addConstraint_RelativeFixed(sys.elements[j+numElements*i],1,sys.elements[j+1+numElements*i],0);
//				//printf("Constraints: %d to %d\n",j+numElements*i,j+1+numElements*i);
//			}
//		}
//
//		for(int i=0;i<numElements-1;i++)
//		{
//			for(int j=0;j<numElements;j++)
//			{
//				element = Element(Node(i*length,-(j+1)*length,0,1,0,0),Node((i+1)*length,-(j+1)*length,0,1,0,0));
//				element.setElasticModulus(EM);
//				element.setDensity(rho);
//				sys.addElement(&element);
//				if(sys.elements.size()%1000==0) printf("Elements added: %d\n",sys.elements.size());
//
//				//printf("Cross Element #%d: (%f, %f, %f) -> (%f, %f, %f)\n",k,i*length,-(j+1)*length,0.0,(i+1)*length,-(j+1)*length,0.0);
//
//
//				sys.addConstraint_RelativeSpherical(sys.elements[j+numElements*i],1,sys.elements[k],0);
//				sys.addConstraint_RelativeSpherical(sys.elements[j+numElements*(i+1)],1,sys.elements[k],1);
//				//printf("Cross Constraint: %d to %d\n",j+numElements*i,k);
//				//printf("Cross Constraint: %d to %d\n",j+numElements*(i+1),k);
//				k++;
//			}
//		}
//	}

	printf("Initializing system (%d beams, %d constraints)... ",sys.elements.size(),sys.constraints.size());
	sys.initializeSystem();
	printf("System Initialized (%d beams, %d constraints, %d equations)!\n",sys.elements.size(),sys.constraints.size(),12*sys.elements.size()+sys.constraints.size());

//	while(sys.getCurrentTime()<=30)
//	{
//		if(sys.getTimeIndex()%100==0) sys.writeToFile();
//		sys.DoTimeStep();
//	}
//	printf("Total time to simulate: %f [s]\n",sys.timeToSimulate);

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowPosition(0,0);
	glutInitWindowSize(1024	,512);
	glutCreateWindow("MAIN");
	glutDisplayFunc(renderSceneAll);
	glutIdleFunc(renderSceneAll);
	glutReshapeFunc(changeSize);
	glutIgnoreKeyRepeat(0);
	glutKeyboardFunc(CallBackKeyboardFunc);
	glutMouseFunc(CallBackMouseFunc);
	glutMotionFunc(CallBackMotionFunc);
	initScene();
	glutMainLoop();

/*
#pragma omp parallel sections
	{
#pragma omp section
		{
			while(true)
			{
//				sys.clearAppliedForces();
//				force.x = -forceMag*sys.p_h[element.getElementIndex()*12+10];
//				force.y = forceMag*sys.p_h[element.getElementIndex()*12+9];
//				force.z = 0;
//				sys.addForce(&element,1,force);
				sys.DoTimeStep();
				//if(sys.timeIndex%100==0) sys.writeToFile();
			}
		}
#pragma omp section
		{
			if(OGL){
				glutInit(&argc, argv);
				glutInitDisplayMode(GLUT_DEPTH | GLUT_DOUBLE | GLUT_RGBA);
				glutInitWindowPosition(0,0);
				glutInitWindowSize(1024	,512);
				glutCreateWindow("MAIN");
				glutDisplayFunc(renderSceneAll);
				glutIdleFunc(renderSceneAll);
				glutReshapeFunc(changeSize);
				glutIgnoreKeyRepeat(0);
				glutKeyboardFunc(CallBackKeyboardFunc);
				glutMouseFunc(CallBackMouseFunc);
				glutMotionFunc(CallBackMotionFunc);
				initScene();
				glutMainLoop();
			}
		}
	}
	*/

	return 0;
}

