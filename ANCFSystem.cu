#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>
#include "include.cuh"
#include "ANCFSystem.cuh"

#include <cusp/io/matrix_market.h>

// linear operator y = A*x (for CUSP)
class stencil: public cusp::linear_operator<double, cusp::device_memory> {
public:
	typedef cusp::linear_operator<double, cusp::device_memory> super;

	int N;
	DeviceView massMatrix;
	DeviceView phiqMatrix;
	DeviceValueArrayView temp;

// constructor
	stencil(int N, DeviceView lhs_mass, DeviceView lhs_phiq,
			DeviceValueArrayView tempVector) :
			super(N, N), N(N) {
		massMatrix = lhs_mass;
		phiqMatrix = lhs_phiq;
		temp = tempVector;
	}

// linear operator y = A*x
	template<typename VectorType1, typename VectorType2>
	void operator()(const VectorType1& x, VectorType2& y) const {
// obtain a raw pointer to device memory
		cusp::multiply(massMatrix, x, temp);
		cusp::multiply(phiqMatrix, x, y);
		cusp::blas::axpy(temp, y, 1);
	}
};

ANCFSystem::ANCFSystem() {

	tol = 1e-7;

	// spike stuff
	//if(useSpike) {
		partitions = 1;
		solverOptions.safeFactorization = true;
		solverOptions.trackReordering = true;
		solverOptions.tolerance = 1e-2*tol;
		//mySpmv = new SpmvFunctor(lhs);
		useSpike = false;
		m_spmv = new MySpmv(lhs_mass,lhs_phiq,lhsVec);
		preconditionerUpdateModulus = 10; // the preconditioner updates every ___ time steps
		preconditionerMaxNewtonIterations = 21; // the preconditioner updates if Newton iterations are greater than ____ iterations
		preconditionerMaxKrylovIterations = 9; // the preconditioner updates if Krylov iterations are greater than ____ iterations
	//}
	// end spike stuff

	this->timeIndex = 0;
	this->time = 0;
	this->h = 0.001;
	alphaHHT = -.1;
	betaHHT = (1 - alphaHHT) * (1 - alphaHHT) * .25;
	gammaHHT = 0.5 - alphaHHT;
	timeToSimulate = 0;
	simTime = 0;
	fullJacobian = 1;

	wt3.push_back(5.0 / 9.0);
	wt3.push_back(8.0 / 9.0);
	wt3.push_back(5.0 / 9.0);

	pt3.push_back(-sqrt(3.0 / 5.0));
	pt3.push_back(0.0);
	pt3.push_back(sqrt(3.0 / 5.0));

	wt5.push_back((322. - 13. * sqrt(70.)) / 900.);
	wt5.push_back((322. + 13. * sqrt(70.)) / 900.);
	wt5.push_back(128. / 225.);
	wt5.push_back((322. + 13. * sqrt(70.)) / 900.);
	wt5.push_back((322. - 13. * sqrt(70.)) / 900.);

	pt5.push_back(-(sqrt(5. + 2. * sqrt(10. / 7.))) / 3.);
	pt5.push_back(-(sqrt(5. - 2. * sqrt(10. / 7.))) / 3.);
	pt5.push_back(0.);
	pt5.push_back((sqrt(5. - 2. * sqrt(10. / 7.))) / 3.);
	pt5.push_back((sqrt(5. + 2. * sqrt(10. / 7.))) / 3.);

	numCollisions = 0;
	numCollisionsSphere = 0;
	numContactPoints = 5;
	coefRestitution = .3;
	frictionCoef = .3;
	fileIndex = 0;

	// set up position files
	char filename1[100];
	char filename2[100];
	char filename3[100];
	sprintf(filename1, "position.dat");
	resultsFile1.open(filename1);
	sprintf(filename2, "energy.dat");
	resultsFile2.open(filename2);
	sprintf(filename3, "reactions.dat");
	resultsFile3.open(filename3);
}

double ANCFSystem::getCurrentTime()
{
	return time;
}
double ANCFSystem::getSimulationTime()
{
	return simTime;
}
double ANCFSystem::getTimeStep()
{
	return h;
}
double ANCFSystem::getTolerance()
{
	return tol;
}
int ANCFSystem::setAlpha_HHT(double alpha)
{
	// should be greater than -.3, usually set to -.1
	alphaHHT = alpha;
	betaHHT = (1 - alphaHHT) * (1 - alphaHHT) * .25;
	gammaHHT = 0.5 - alphaHHT;
	return 0;
}
int ANCFSystem::setSimulationTime(double simTime)
{
	this->simTime = simTime;
	return 0;
}
int ANCFSystem::setTimeStep(double h)
{
	this->h = h;
	return 0;
}
int ANCFSystem::setTolerance(double tolerance)
{
	this->tol = tolerance;
	solverOptions.tolerance = tolerance*1e-2;
	return 0;
}
int ANCFSystem::getTimeIndex()
{
	return this->timeIndex;
}
int ANCFSystem::setSolverTolerance(double tolerance)
{
	solverOptions.tolerance = tolerance;
	return 0;
}
int ANCFSystem::setPartitions(int partitions)
{
	this->partitions = partitions;
	return 0;
}

int ANCFSystem::addParticle(Particle* particle)
{
	//add the element
	particle->setParticleIndex(particles.size());
	this->particles.push_back(*particle);

	MaterialParticle material;
	material.E = particle->getElasticModulus();
	material.nu = particle->getNu();
	material.mass = particle->getMass();
	material.massInverse = 1.0/particle->getMass();
	material.r = particle->getRadius();
	material.numContactPoints = 1;
	this->pMaterials_h.push_back(material);

	// update p
	float3 pos0 = particle->getInitialPosition();
	pParticle_h.push_back(pos0.x);
	pParticle_h.push_back(pos0.y);
	pParticle_h.push_back(pos0.z);

	// update v
	float3 vel0 = particle->getInitialVelocity();
	vParticle_h.push_back(vel0.x);
	vParticle_h.push_back(vel0.y);
	vParticle_h.push_back(vel0.z);

	for(int i=0;i<3;i++)
	{
		aParticle_h.push_back(0.0);
		fParticle_h.push_back(0.0);
	}

	return particles.size();
}

int ANCFSystem::addElement(Element* element)
{
	//add the element
	element->setElementIndex(elements.size());
	this->elements.push_back(*element);

	Material material;
	material.E = element->getElasticModulus();
	material.l = element->getLength_l();
	material.nu = element->getNu();
	material.rho = element->getDensity();
	material.r = element->getRadius();
	material.numContactPoints = numContactPoints;
	this->materials.push_back(material);

	// update p
	Node node = element->getNode0();
	p_h.push_back(node.x);
	p_h.push_back(node.y);
	p_h.push_back(node.z);
	p_h.push_back(node.dx1);
	p_h.push_back(node.dy1);
	p_h.push_back(node.dz1);
	node = element->getNode1();
	p_h.push_back(node.x);
	p_h.push_back(node.y);
	p_h.push_back(node.z);
	p_h.push_back(node.dx1);
	p_h.push_back(node.dy1);
	p_h.push_back(node.dz1);

	for(int i=0;i<12;i++)
	{
		e_h.push_back(0.0);
		v_h.push_back(0.0);
		a_h.push_back(0.0);
		anew_h.push_back(0.0);
		fint_h.push_back(0.0);
		fcon_h.push_back(0.0);
		fapp_h.push_back(0.0);
		phiqlam_h.push_back(0.0);
		delta_h.push_back(0.0);
		strainDerivative_h.push_back(0.0);
	}
	strain_h.push_back(0.0);

	for(int i=0;i<4;i++)
	{
		Sx_h.push_back(0.0);
		Sxx_h.push_back(0.0);
	}

	//update other vectors (no initial velocity or acceleration)
	double r = element->getRadius();
	double a = element->getLength_l();
	double rho = element->getDensity();
	double A = PI*r*r;
		
	// update external force vector (gravity)
	fext_h.push_back(rho * A * a * GRAVITYx / 0.2e1);
	fext_h.push_back(rho * A * a * GRAVITYy / 0.2e1);
	fext_h.push_back(rho * A * a * GRAVITYz / 0.2e1);
	fext_h.push_back(rho * A * a * a * GRAVITYx / 0.12e2);
	fext_h.push_back(rho * A * a * a * GRAVITYy / 0.12e2);
	fext_h.push_back(rho * A * a * a * GRAVITYz / 0.12e2);
	fext_h.push_back(rho * A * a * GRAVITYx / 0.2e1);
	fext_h.push_back(rho * A * a * GRAVITYy / 0.2e1);
	fext_h.push_back(rho * A * a * GRAVITYz / 0.2e1);
	fext_h.push_back(-rho * A * a * a * GRAVITYx / 0.12e2);
	fext_h.push_back(-rho * A * a * a * GRAVITYy / 0.12e2);
	fext_h.push_back(-rho * A * a * a * GRAVITYz / 0.12e2);

	for(int i=0;i<12;i++)
	{
		for(int j=0;j<12;j++)
		{
			lhsI_h.push_back(i+12*(elements.size()-1));
			lhsJ_h.push_back(j+12*(elements.size()-1));
			lhs_h.push_back(0.0);
		}
	}

	return elements.size();
}

int ANCFSystem::addForce(Element* element, double xi, float3 force)
{
	int index = element->getElementIndex();
	int l = element->getLength_l();

	//fapp_h = fapp_d;

	fapp_h[ 0+12*index] += (1 - 3 * xi * xi + 2 * pow( xi, 3)) * force.x;
	fapp_h[ 1+12*index] += (1 - 3 * xi * xi + 2 * pow( xi, 3)) * force.y;
	fapp_h[ 2+12*index] += (1 - 3 * xi * xi + 2 * pow( xi, 3)) * force.z;
	fapp_h[ 3+12*index] += l * (xi - 2 * xi * xi + pow( xi, 3)) * force.x;
	fapp_h[ 4+12*index] += l * (xi - 2 * xi * xi + pow( xi, 3)) * force.y;
	fapp_h[ 5+12*index] += l * (xi - 2 * xi * xi + pow( xi, 3)) * force.z;
	fapp_h[ 6+12*index] += (3 * xi * xi - 2 * pow( xi, 3)) * force.x;
	fapp_h[ 7+12*index] += (3 * xi * xi - 2 * pow( xi, 3)) * force.y;
	fapp_h[ 8+12*index] += (3 * xi * xi - 2 * pow( xi, 3)) * force.z;
	fapp_h[ 9+12*index] += l * (-xi * xi + pow( xi, 3)) * force.x;
	fapp_h[10+12*index] += l * (-xi * xi + pow( xi, 3)) * force.y;
	fapp_h[11+12*index] += l * (-xi * xi + pow( xi, 3)) * force.z;

	fapp_d = fapp_h;

	return 0;
}

int ANCFSystem::clearAppliedForces()
{
	thrust::fill(fapp_d.begin(),fapp_d.end(),0.0); //Clear internal forces
	return 0;
}

int ANCFSystem::updatePhiq() // used in Newton iteration, nice to keep it separate (but not memory efficient) - only needs to be done once (linear constraints)
{
	for(int i=0;i<constraints.size();i++)
	{
		Constraint constraint = constraints[i];

		phiqJ_h.push_back(i);
		phiqI_h.push_back(constraint.dofLoc.x);
		phiq_h.push_back(1.0);

		if(constraint.nodeNum2!=-1)
		{
			phiqJ_h.push_back(i);
			phiqI_h.push_back(constraint.dofLoc.y);
			phiq_h.push_back(-1.0);
		}
	}
	phiqI_d = phiqI_h;
	phiqJ_d = phiqJ_h;
	phiq_d = phiq_h;

	thrust::device_ptr<int> wrapped_device_I(CASTI1(phiqI_d));
	DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + phiqI_d.size());

	thrust::device_ptr<int> wrapped_device_J(CASTI1(phiqJ_d));
	DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + phiqJ_d.size());

	thrust::device_ptr<double> wrapped_device_V(CASTD1(phiq_d));
	DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + phiq_d.size());

	phiq = DeviceView(12*elements.size(), constraints.size(), phiq_d.size(), row_indices, column_indices, values);
	phiq.sort_by_row();

	return 0;
}

__global__ void calculateRHSlower(double* phi, double* p, double* phi0, double factor, int2* constraintPairs, int numConstraints)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if(i<numConstraints)
	{
		int2 constraintPair = constraintPairs[i];
		if(constraintPair.y == -1)
		{
			phi[i] = factor*(p[constraintPair.x]-phi0[i]);
		}
		else
		{
			phi[i] = factor*(p[constraintPair.x]-p[constraintPair.y]-phi0[i]);
		}
		__syncthreads();
	}
}

int ANCFSystem::updatePhi()
{
	calculateRHSlower<<<dimGridConstraint,dimBlockConstraint>>>(CASTD1(phi_d), CASTD1(pnew_d), CASTD1(phi0_d), 1.0/(betaHHT*h*h), CASTI2(constraintPairs_d), constraints.size());

	return 0;
}

__global__ void updateParticleDynamics_GPU(double h, double* a, double* v, double* p, double* f, MaterialParticle* materials, int numParticles)
{
	int i = threadIdx.x+blockIdx.x*blockDim.x;

	if(i<numParticles)
	{
		a = &a[3*i];
		v = &v[3*i];
		p = &p[3*i];
		f = &f[3*i];
		MaterialParticle material = materials[i];

		a[0] = material.massInverse*f[0]+GRAVITYx;
		a[1] = material.massInverse*f[1]+GRAVITYy;
		a[2] = material.massInverse*f[2]+GRAVITYz;

		for(int j=0;j<3;j++)
		{
			v[j] += h*a[j];
			p[j] += h*v[j];
		}
	}
}

int ANCFSystem::updateParticleDynamics()
{
	updateParticleDynamics_GPU<<<dimGridParticles,dimBlockParticles>>>(h,CASTD1(aParticle_d), CASTD1(vParticle_d), CASTD1(pParticle_d), CASTD1(fParticle_d), CASTMP(pMaterials_d), particles.size());

	return 0;
}

int ANCFSystem::calculateInitialPhi()
{
	for(int i=0;i<constraints.size();i++) phi0_h.push_back(0);
	for(int i=0;i<constraints.size();i++)
	{
		Constraint constraint = constraints[i];

		if(constraint.nodeNum2 == -1)
		{
			phi0_h[i] = p_h[constraint.dofLoc.x];
		}
		else
		{
			phi0_h[i] = p_h[constraint.dofLoc.x]-p_h[constraint.dofLoc.y];
		}
	}

	return 0;
}

int ANCFSystem::initializeDevice()
{
	pMaterials_d = pMaterials_h;
	pParticle_d = pParticle_h;
	vParticle_d = vParticle_h;
	aParticle_d = aParticle_h;
	fParticle_d = fParticle_h;

	materials_d = materials;
	strainDerivative_d = strainDerivative_h;
	curvatureDerivative_d = strainDerivative_h;
	strain_d = strain_h;
	Sx_d = Sx_h;
	Sxx_d = Sxx_h;

	e_d = e_h;
	p_d = p_h;
	v_d = v_h;
	a_d = a_h;
	pnew_d = p_h;
	vnew_d = v_h;
	anew_d = anew_h;
	fext_d = fext_h;
	fint_d = fint_h;
	fapp_d = fapp_h;
	fcon_d = fcon_h;
	phi_d = phi_h;
	phi0_d = phi0_h;
	phiqlam_d = phiqlam_h;
	delta_d = delta_h;
	constraintPairs_d = constraintPairs_h;
	lhsVec_d = anew_h;

	lhsI_d = lhsI_h;
	lhsJ_d = lhsJ_h;
	lhs_d = lhs_h;

	constraintsI_d = constraintsI_h;
	constraintsJ_d = constraintsJ_h;
	constraints_d = constraints_h;

	thrust::device_ptr<double> wrapped_device_e(CASTD1(e_d));
	thrust::device_ptr<double> wrapped_device_p(CASTD1(p_d));
	thrust::device_ptr<double> wrapped_device_v(CASTD1(v_d));
	thrust::device_ptr<double> wrapped_device_a(CASTD1(a_d));
	thrust::device_ptr<double> wrapped_device_pnew(CASTD1(pnew_d));
	thrust::device_ptr<double> wrapped_device_vnew(CASTD1(vnew_d));
	thrust::device_ptr<double> wrapped_device_anew(CASTD1(anew_d));
	thrust::device_ptr<double> wrapped_device_fext(CASTD1(fext_d));
	thrust::device_ptr<double> wrapped_device_fint(CASTD1(fint_d));
	thrust::device_ptr<double> wrapped_device_fapp(CASTD1(fapp_d));
	thrust::device_ptr<double> wrapped_device_fcon(CASTD1(fcon_d));
	thrust::device_ptr<double> wrapped_device_phi(CASTD1(phi_d));
	thrust::device_ptr<double> wrapped_device_phi0(CASTD1(phi0_d));
	thrust::device_ptr<double> wrapped_device_phiqlam(CASTD1(phiqlam_d));
	thrust::device_ptr<double> wrapped_device_delta(CASTD1(delta_d));
	thrust::device_ptr<double> wrapped_device_lhsVec(CASTD1(lhsVec_d));

	eAll = DeviceValueArrayView(wrapped_device_e, wrapped_device_e + e_d.size());
	eTop = DeviceValueArrayView(wrapped_device_e, wrapped_device_e + 12*elements.size());
	eBottom = DeviceValueArrayView(wrapped_device_e + 12*elements.size(), wrapped_device_e + e_d.size());
	p = DeviceValueArrayView(wrapped_device_p, wrapped_device_p + p_d.size());
	v = DeviceValueArrayView(wrapped_device_v, wrapped_device_v + v_d.size());
	a = DeviceValueArrayView(wrapped_device_a, wrapped_device_a + a_d.size());
	pnew = DeviceValueArrayView(wrapped_device_pnew, wrapped_device_pnew + pnew_d.size());
	vnew = DeviceValueArrayView(wrapped_device_vnew, wrapped_device_vnew + vnew_d.size());
	anewAll = DeviceValueArrayView(wrapped_device_anew, wrapped_device_anew + anew_d.size());
	anew = DeviceValueArrayView(wrapped_device_anew, wrapped_device_anew + 12*elements.size());
	lambda = DeviceValueArrayView(wrapped_device_anew + 12*elements.size(), wrapped_device_anew + anew_d.size());
	fext = DeviceValueArrayView(wrapped_device_fext, wrapped_device_fext + fext_d.size());
	fint = DeviceValueArrayView(wrapped_device_fint, wrapped_device_fint + fint_d.size());
	fapp = DeviceValueArrayView(wrapped_device_fapp, wrapped_device_fapp + fapp_d.size());
	fcon = DeviceValueArrayView(wrapped_device_fcon, wrapped_device_fcon + fcon_d.size());
	phi = DeviceValueArrayView(wrapped_device_phi, wrapped_device_phi + phi_d.size());
	phi0 = DeviceValueArrayView(wrapped_device_phi0, wrapped_device_phi0 + phi0_d.size());
	phiqlam = DeviceValueArrayView(wrapped_device_phiqlam, wrapped_device_phiqlam + phiqlam_d.size());
	delta = DeviceValueArrayView(wrapped_device_delta, wrapped_device_delta + delta_d.size());
	lhsVec = DeviceValueArrayView(wrapped_device_lhsVec, wrapped_device_lhsVec + lhsVec_d.size());

	// create lhs matrix using cusp library (shouldn't change)
	thrust::device_ptr<int> wrapped_device_I(CASTI1(lhsI_d));
	DeviceIndexArrayView row_indices = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + lhsI_d.size());

	thrust::device_ptr<int> wrapped_device_J(CASTI1(lhsJ_d));
	DeviceIndexArrayView column_indices = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + lhsJ_d.size());

	thrust::device_ptr<double> wrapped_device_V(CASTD1(lhs_d));
	DeviceValueArrayView values = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + lhs_d.size());

	lhs = DeviceView( anew_d.size(), anew_d.size(), lhs_d.size(), row_indices, column_indices, values);
	// end create lhs matrix

	// create the view to the mass block of the lhs matrix
	DeviceIndexArrayView row_indices_mass = DeviceIndexArrayView(wrapped_device_I, wrapped_device_I + 12*12*elements.size());
	DeviceIndexArrayView column_indices_mass = DeviceIndexArrayView(wrapped_device_J, wrapped_device_J + 12*12*elements.size());
	DeviceValueArrayView values_mass = DeviceValueArrayView(wrapped_device_V, wrapped_device_V + 12*12*elements.size());
	lhs_mass = DeviceView( anew_d.size(), anew_d.size(), 12*12*elements.size(), row_indices_mass, column_indices_mass, values_mass);
	// end create the view to the mass block of the lhs matrix

	// create the view to the mass block of the lhs matrix
	DeviceIndexArrayView row_indices_phiq = DeviceIndexArrayView(wrapped_device_I + 12*12*elements.size(), wrapped_device_I + lhsI_d.size());
	DeviceIndexArrayView column_indices_phiq = DeviceIndexArrayView(wrapped_device_J + 12*12*elements.size(), wrapped_device_J + lhsJ_d.size());
	DeviceValueArrayView values_phiq = DeviceValueArrayView(wrapped_device_V + 12*12*elements.size(), wrapped_device_V + lhs_d.size());
	lhs_phiq = DeviceView( anew_d.size(), anew_d.size(), lhs_d.size()-12*12*elements.size(), row_indices_phiq, column_indices_phiq, values_phiq);
	lhs_phiq.sort_by_row(); // MUST BE SORTED FOR SPMV TO WORK CORRECTLY
	// end create the view to the mass block of the lhs matrix

	dimBlockConstraint.x = BLOCKDIMCONSTRAINT;
	dimGridConstraint.x = static_cast<int>(ceil((static_cast<double>(constraints.size()))/(static_cast<double>(BLOCKDIMCONSTRAINT))));

	dimBlockElement.x = BLOCKDIMELEMENT;
	dimGridElement.x = (int)ceil(((double)(elements.size()))/((double)BLOCKDIMELEMENT));

	dimBlockParticles.x = BLOCKDIMELEMENT;
	dimGridParticles.x = (int)ceil(((double)(particles.size()))/((double)BLOCKDIMELEMENT));

	dimBlockCollision.x = BLOCKDIMCOLLISION;
	dimGridCollision.x = (int)ceil(((double)(particles.size()))/((double)BLOCKDIMCOLLISION));

	return 0;
}

int ANCFSystem::initializeSystem()
{
	ANCFSystem::updatePhiq();
	ANCFSystem::calculateInitialPhi();

	for(int i=0;i<constraints.size();i++)
	{
		delta_h.push_back(0);
		e_h.push_back(0);
		anew_h.push_back(0);
		phi_h.push_back(0);
		constraintPairs_h.push_back(constraints[i].dofLoc);
	}

	// join phi_q to lhs
	for(int i=0;i<constraints.size();i++)
	{
		Constraint constraint = constraints[i];
		lhsI_h.push_back(i+12*elements.size());
		lhsJ_h.push_back(constraint.dofLoc.x);
		lhs_h.push_back(1.0);

		if(constraint.nodeNum2!=-1)
		{
			lhsI_h.push_back(i+12*elements.size());
			lhsJ_h.push_back(constraint.dofLoc.y);
			lhs_h.push_back(-1.0);
		}
	}

	// join phi_q' to lhs
	for(int i=0;i<constraints.size();i++)
	{
		Constraint constraint = constraints[i];
		lhsJ_h.push_back(i+12*elements.size());
		lhsI_h.push_back(constraint.dofLoc.x);
		lhs_h.push_back(1.0);

		if(constraint.nodeNum2!=-1)
		{
			lhsJ_h.push_back(i+12*elements.size());
			lhsI_h.push_back(constraint.dofLoc.y);
			lhs_h.push_back(-1.0);
		}
	}

	// Get constraints
	for(int i=0;i<constraints.size();i++)
	{
		Constraint constraint = constraints[i];
		constraintsI_h.push_back(i+12*elements.size());
		constraintsJ_h.push_back(constraint.dofLoc.x);
		constraints_h.push_back(1.0);

		if(constraint.nodeNum2!=-1)
		{
			constraintsI_h.push_back(i+12*elements.size());
			constraintsJ_h.push_back(constraint.dofLoc.y);
			constraints_h.push_back(-1.0);
		}
	}

	// join phi_q' to lhs
	for(int i=0;i<constraints.size();i++)
	{
		Constraint constraint = constraints[i];
		constraintsJ_h.push_back(i+12*elements.size());
		constraintsI_h.push_back(constraint.dofLoc.x);
		constraints_h.push_back(1.0);

		if(constraint.nodeNum2!=-1)
		{
			constraintsJ_h.push_back(i+12*elements.size());
			constraintsI_h.push_back(constraint.dofLoc.y);
			constraints_h.push_back(-1.0);
		}
	}

	initializeDevice();
	//ANCFSystem::initializeBoundingBoxes_CPU();
	//detector.updateBoundingBoxes(aabb_data_d);
	//detector.setBoundingBoxPointer(&aabb_data_d);
	//detector.detectPossibleCollisions();

	ANCFSystem::resetLeftHandSideMatrix();
	ANCFSystem::updateInternalForces();

	//cusp::blas::axpy(fint,eTop,-1);
	cusp::blas::axpby(fext,fint,eTop,1,-1);

	// spike stuff
	if(useSpike) {
		mySolver = new SpikeSolver(partitions,solverOptions);
		mySolver->setup(lhs);
	}
	// end spike stuff

	// cusp stuff
	//M = new cusp::precond::scaled_bridson_ainv<double, cusp::device_memory>(lhs, .1);
	// end cusp stuff

	char filename[100];
	sprintf(filename, "./lhs.txt");
	cusp::io::write_matrix_market_file(lhs, filename);

	//cusp::print(lhs);
	//cusp::print(eAll);
	//cin.get();

	//lhs.sort_by_row();
	//cusp::blas::fill(delta,0);
	if(!useSpike)
	{
		// SOLVE USING CUSP
		stencil lhsStencil(anewAll.size(), lhs_mass, lhs_phiq, lhsVec);

		cusp::default_monitor<double> monitor(eAll, 1000, 1e-2*tol);

		//cusp::precond::diagonal<double, cusp::device_memory> M(lhs);

		// solve the linear system A * x = b with the Bi-Conjugate Gradient - Stable method
		cusp::krylov::cg(lhsStencil, delta, eAll, monitor);//, M);

		//cout << "Success: " << monitor.converged() << " Iterations: "
		//		<< monitor.iteration_count() << " relResidualNorm: "
		//		<< monitor.relative_tolerance() << " norm_d: ";
	}

	if(useSpike)
	{
		bool success = mySolver->solve(*m_spmv,eAll,anewAll);
		spike::Stats stats = mySolver->getStats();
	}

	//cusp::print(delta);
	//cin.get();

	//cusp::copy(delta,anewAll);
	cusp::copy(anew,a);
	cusp::copy(v,vnew);
	cusp::copy(p,pnew);

	//ANCFSystem::updateParticleDynamics();

	return 0;
}

int ANCFSystem::DoTimeStep()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	double norm_e=1;
	double norm_d=1;
	double norm_d_0 = 1;
	int it = 0;

	//ANCFSystem::updateParticleDynamics();
	stepKrylovIterations = 0;

	// update q and q_dot for initial guess
	cusp::blas::axpbypcz(p,v,a,pnew,1,h,.5*h*h);
	cusp::blas::axpby(v,a,vnew,1,h);

	if(useSpike&&timeIndex%preconditionerUpdateModulus==0)
	{
			mySolver->update(lhs.values);
			//printf("Preconditioner updated!\n");
	}

	while(norm_d>tol&&it<100)//while(norm_e>tol&&norm_d>tol)
	{
		it++;

		ANCFSystem::updatePhi();
		cusp::multiply(phiq,lambda,phiqlam);
		ANCFSystem::resetLeftHandSideMatrix();
		cusp::multiply(lhs_mass,anew,eTop); //cusp::multiply(mass,anew,eTop);
		ANCFSystem::updateInternalForces();
		cusp::blas::axpbypcz(eTop,fapp,fint,eTop,1,-1,1);
		cusp::blas::axpby(eTop,fext,eTop,1,-1);
		cusp::blas::axpy(phiqlam,eTop,1);
		cusp::blas::copy(phi,eBottom);

//		cusp::print(fapp);
//		cusp::print(fint);
//		cin.get();

		// SOLVE THE LINEAR SYSTEM

		cusp::blas::fill(delta, 0);
		if(!useSpike)
		{
			// SOLVE USING CUSP
			stencil lhsStencil(anewAll.size(), lhs_mass, lhs_phiq, lhsVec);

			cusp::default_monitor<double> monitor(eAll, 1000, 1e-2*tol);

			//cusp::precond::diagonal<double, cusp::device_memory> M(lhs);

			// solve the linear system A * x = b with the Bi-Conjugate Gradient - Stable method
			cusp::krylov::cg(lhsStencil, delta, eAll, monitor);//, M);
			stepKrylovIterations += monitor.iteration_count();

			//cout << "Success: " << monitor.converged() << " Iterations: "
			//		<< monitor.iteration_count() << " relResidualNorm: "
			//		<< monitor.relative_tolerance() << " norm_d: ";
		}

		if(useSpike)
		{
			//SOLVE USING SPIKE
			bool success = mySolver->solve(*m_spmv,eAll,delta);

			spike::Stats stats = mySolver->getStats();
			if(useSpike&&stats.numIterations>preconditionerMaxKrylovIterations)
			{
				mySolver->update(lhs.values);
				//printf("Preconditioner updated!\n");
			}
			stepKrylovIterations += stats.numIterations;

			//cout << "Success: " << success << " Iterations: "
			//		<< stats.numIterations << " relResidualNorm: "
			//		<< stats.relResidualNorm << " norm_d: ";
		}
		// END SOLVE THE LINEAR SYSTEM

//		if(timeIndex%100==0)
//		{
//			char filename[100];
//			sprintf(filename, "./intForce%d.dat", timeIndex/100);
//			cusp::io::write_matrix_market_file(fint, filename);
//		}

		// update anew
		cusp::blas::axpy(delta,anewAll,-1);

		// update vnew
		cusp::blas::axpbypcz(v,a,anew,vnew,1,h*(1-gammaHHT),h*gammaHHT);

		// update pnew
		cusp::blas::axpbypcz(v,a,anew,pnew,h,h*h*.5*(1-2*betaHHT),h*h*.5*2*betaHHT);
		cusp::blas::axpy(p,pnew,1);

		// get norms
		//norm_e = cusp::blas::nrm2(eAll)/pow((double)elements.size(),2);
		if(it==1) norm_d_0 = cusp::blas::nrm2(delta);
		norm_d = cusp::blas::nrm2(delta)/norm_d_0;
		//out << norm_d << endl;
		//cout << norm_e << " " << norm_d << endl;
	}

	if(useSpike&&it>preconditionerMaxNewtonIterations)
	{
		mySolver->update(lhs.values);
		//printf("Preconditioner updated!\n");
	}

	cusp::copy(anew,a);
	cusp::copy(vnew,v);
	cusp::copy(pnew,p);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime , start, stop);

	timeToSimulate+=elapsedTime/1000.0;

	p_h = p_d;
	//v_h = v_d;
	//pParticle_h = pParticle_d;
	//vParticle_h = vParticle_d;

	//printf("Time: %f (it = %d, PTA pos = (%f, %.13f, %f)\n",this->getCurrentTime(),it,getXYZPosition(elements.size()-1,1).x,getXYZPosition(elements.size()-1,1).y,getXYZPosition(elements.size()-1,1).z);
	//printf("Time: %f (Simulation time = %f ms, it = %d)\n",this->getCurrentTime(), elapsedTime,it);
	printf("%f, %f, %d, \n",this->getCurrentTime(), elapsedTime, it);
	//printf("Pos = (%f, %.13f, %f)\n",getXYZPosition(elements.size()-1,1).x,getXYZPosition(elements.size()-1,1).y,getXYZPosition(elements.size()-1,1).z);
	//printf("%f\n",getXYZPosition(elements.size()-1,1).y);
	
	stepTime = elapsedTime;
	stepNewtonIterations = it;
	

	time+=h;
	timeIndex++;

	return 0;
}

float3 ANCFSystem::getXYZPosition(int elementIndex, double xi)
{
	double a = elements[elementIndex].getLength_l();
	double* p = CASTD1(p_h);
	p = &p[12*elementIndex];
	float3 pos;

	pos.x = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[0] + a * (xi - 2 * xi * xi + pow(xi, 3)) * p[3] + (3 * xi * xi - 2 * pow(xi, 3)) * p[6] + a * (-xi * xi + pow(xi, 3)) * p[9];
	pos.y = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[1] + a * (xi - 2 * xi * xi + pow(xi, 3)) * p[4] + (3 * xi * xi - 2 * pow(xi, 3)) * p[7] + a * (-xi * xi + pow(xi, 3)) * p[10];
	pos.z = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[2] + a * (xi - 2 * xi * xi + pow(xi, 3)) * p[5] + (3 * xi * xi - 2 * pow(xi, 3)) * p[8] + a * (-xi * xi + pow(xi, 3)) * p[11];

	return pos;
}

float3 ANCFSystem::getXYZVelocity(int elementIndex, double xi)
{
	double a = elements[elementIndex].getLength_l();
	double* p = CASTD1(v_h);
	p = &p[12*elementIndex];
	float3 pos;

	pos.x = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[0] + a * (xi - 2 * xi * xi + pow(xi, 3)) * p[3] + (3 * xi * xi - 2 * pow(xi, 3)) * p[6] + a * (-xi * xi + pow(xi, 3)) * p[9];
	pos.y = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[1] + a * (xi - 2 * xi * xi + pow(xi, 3)) * p[4] + (3 * xi * xi - 2 * pow(xi, 3)) * p[7] + a * (-xi * xi + pow(xi, 3)) * p[10];
	pos.z = (1 - 3 * xi * xi + 2 * pow(xi, 3)) * p[2] + a * (xi - 2 * xi * xi + pow(xi, 3)) * p[5] + (3 * xi * xi - 2 * pow(xi, 3)) * p[8] + a * (-xi * xi + pow(xi, 3)) * p[11];

	return pos;
}


float3 ANCFSystem::getXYZPositionParticle(int index)
{
	return make_float3(pParticle_h[3*index],pParticle_h[3*index+1],pParticle_h[3*index+2]);
}

float3 ANCFSystem::getXYZVelocityParticle(int index)
{
	return make_float3(vParticle_h[3*index],vParticle_h[3*index+1],vParticle_h[3*index+2]);
}

int ANCFSystem::saveLHS()
{
	char filename[100];
	posFile.open("../lhs.dat");
	posFile << "symmetric" << endl;
	posFile << anew_h.size() << " " << anew_h.size() << " " << lhsI_h.size() << endl;
	for(int i=0;i<lhsI_h.size();i++)
	{
		posFile << lhsI_h[i] << " " << lhsJ_h[i] << " " << lhs_h[i] << endl;
	}
	posFile.close();

	return 0;
}

int ANCFSystem::writeToFile(string fileName)
{
	//char filename1[100];
	//sprintf(filename1, "./posData/lhs%d.dat", fileIndex);
	//cusp::io::write_matrix_market_file(lhs, filename1);

	posFile.open(fileName.c_str());
	p_h = p_d;
	double* posAll = CASTD1(p_h);
	double* pos;
	float3 posPart;
	double l;
	double r;
	posFile << elements.size()<<  ","  << endl;
//	for(int i=0;i<particles.size();i++)
//	{
//		r = particles[i].getRadius();
//		posPart = getXYZPositionParticle(i);
//		posFile << r << ", " << posPart.x << ", " << posPart.y << ", " << posPart.z << "," << endl;
//	}
	for(int i=0;i<elements.size();i++)
	{
		l = elements[i].getLength_l();
		r = elements[i].getRadius();
		pos = &posAll[12*i];
		posFile << r << "," << l;
		for(int i=0;i<12;i++) posFile << "," << pos[i];
		posFile << ","<< endl;
	}
	posFile.close();

	return 0;
}
